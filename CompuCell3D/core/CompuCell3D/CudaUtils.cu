#include <iostream>
#include <hip/hip_runtime.h>
#include "CudaUtils.cuh"

using namespace std;

void chooseCudaDevice()
{
  // TODO: only call this once
  // Error or do nothing?
  cout << "Selecting the fastest GPU device...\n";
  int num_devices, device;
  checkCudaErrors(hipGetDeviceCount(&num_devices));
  if (num_devices > 1)
  {
    int max_multiprocessors = 0, max_device = 0;
    for (device = 0; device < num_devices; device++)
    {
      hipDeviceProp_t properties;
      checkCudaErrors(hipGetDeviceProperties(&properties, device));
      if (max_multiprocessors < properties.multiProcessorCount)
      {
        max_multiprocessors = properties.multiProcessorCount;
        max_device = device;
      }
    }
    hipDeviceProp_t properties;
    checkCudaErrors(hipGetDeviceProperties(&properties, max_device));
    cout << "GPU device " << max_device << " selected; GPU device name: " << properties.name << endl;
    checkCudaErrors(hipSetDevice(max_device));
  }
  else
  {
    cout << "Only one GPU device available, will use it (#0)\n";
    hipDeviceProp_t properties;
    int device = 0;
    // checkCudaErrors(hipGetDeviceProperties(&properties, device));
    // cout << "GPU device name: " << properties.name << endl;
    // cout << "Device Number: " << 0 << endl;
    // cout << "  Memory Clock Rate (KHz): " << properties.memoryClockRate << endl;
    // cout << "  Memory Bus Width (bits): " << properties.memoryBusWidth << endl;
    // cout << "  Peak Memory Bandwidth (GB/s): " << 2.0*properties.memoryClockRate*(properties.memoryBusWidth/8)/1.0e6 << endl;
    // cout << "  Compute capability: " << properties.major << "." << properties.minor << endl;
    checkCudaErrors(hipSetDevice(device));
  }
}
