#include "hip/hip_runtime.h"
#include <math.h>
#include <CompuCell3D/Boundary/BoundaryStrategy.h>
#include <CompuCell3D/CC3DExceptions.h>

#include "Dim3D.h"
#include "Field3DImpl.cuh"
#include "Field3D.h"
// #include <CompuCell3D/steppables/PDESolvers/CUDA/CUDAUtilsHeader.h>
#include <CompuCell3D/CudaUtils.cuh>
#include <hip/hip_runtime.h>

using namespace CompuCell3D;

/**
 * @param dim The field dimensions
 * @param initialValue The initial value of all data elements in the field.
 */
template <class T>
Field3DImpl<T>::Field3DImpl(const Dim3D dim, const T &initialValue) : dim(dim), field(0), initialValue(initialValue)
{

  if (dim.x == 0 && dim.y == 0 && dim.z == 0)
    throw CC3DException("Field3D cannot have a 0 dimension!!!");

  // Check that the dimensions are not too large.
  if (log((double)dim.x) / log(2.0) + log((double)dim.y) / log(2.0) + log((double)dim.z) / log(2.0) >
      sizeof(int) * 8)
    throw CC3DException("Field3D dimensions too large!!!");

  // Allocate and initialize the field
  len = dim.x * dim.y * dim.z;
  // field = new T[len];
  checkCudaErrors(hipMallocManaged(&field, len * sizeof(T)));
  for (unsigned int i = 0; i < len; i++)
    field[i] = initialValue;
}

template <class T>
Field3DImpl<T>::~Field3DImpl()
{
  if (field)
  {
    // delete[] field;
    checkCudaErrors(hipFree(&field));
    field = 0;
  }
}

template <class T>
T *Field3DImpl<T>::getPtr()
{
  return field;
}

template <class T>
void Field3DImpl<T>::set(const Point3D &pt, const T value)
{
  if (!isValid(pt))
    throw CC3DException("set() point out of range!");
  field[PT2IDX(pt)] = value;
}

template <class T>
void Field3DImpl<T>::resizeAndShift(const Dim3D theDim, Dim3D shiftVec = Dim3D())
{
  // T *field2 = new T[theDim.x * theDim.y * theDim.z];
  T *field2;
  checkCudaErrors(hipMallocManaged(&field2, theDim.x * theDim.y * theDim.z * sizeof(T)));

  // first initialize the lattice with initial value
  for (long int i = 0; i < theDim.x * theDim.y * theDim.z; ++i)
    field2[i] = initialValue;

  // then  copy old field
  for (int x = 0; x < theDim.x; x++)
    for (int y = 0; y < theDim.y; y++)
      for (int z = 0; z < theDim.z; z++)
        if ((x - shiftVec.x >= 0) && (x - shiftVec.x < dim.x) && (y - shiftVec.y >= 0) &&
            (y - shiftVec.y < dim.y) && (z - shiftVec.z >= 0) && (z - shiftVec.z < dim.z))
        {
          field2[x + ((y + (z * theDim.y)) * theDim.x)] = getQuick(
              Point3D(x - shiftVec.x, y - shiftVec.y, z - shiftVec.z));
        }

  // delete[]  field;
  checkCudaErrors(hipFree(&field));
  field = field2;
  dim = theDim;

  // Set dimension for the Boundary Strategy
  BoundaryStrategy::getInstance()->setDim(dim);
}

template <class T>
void Field3DImpl<T>::setDim(const Dim3D theDim)
{
  this->resizeAndShift(theDim);
}

template <class T>
T Field3DImpl<T>::getQuick(const Point3D &pt) const
{

  // return field[PT2IDX(pt)];
  return (isValid(pt) ? field[PT2IDX(pt)] : initialValue);
}

template <class T>
void Field3DImpl<T>::setQuick(const Point3D &pt, const T _value)
{

  field[PT2IDX(pt)] = _value;
}

template <class T>
T Field3DImpl<T>::get(const Point3D &pt) const
{

  return (isValid(pt) ? field[PT2IDX(pt)] : initialValue);
}

template <class T>
T Field3DImpl<T>::getByIndex(long _offset) const
{

  return (((0 <= _offset) && (_offset < len)) ? field[_offset] : initialValue);
}

template <class T>
void Field3DImpl<T>::setByIndex(long _offset, const T _value)
{
  if ((0 <= _offset) && (_offset < len))
    field[_offset] = _value;
}

template <class T>
Dim3D Field3DImpl<T>::getDim() const { return dim; }

template <class T>
bool Field3DImpl<T>::isValid(const Point3D &pt) const
{
  return (0 <= pt.x && pt.x < dim.x &&
          0 <= pt.y && pt.y < dim.y &&
          0 <= pt.z && pt.z < dim.z);
}
